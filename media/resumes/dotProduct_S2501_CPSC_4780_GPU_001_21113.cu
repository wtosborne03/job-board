
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

const int threads_per_block = 512;

// Forward function declarations
float CPU_big_dot(float *A, float *B, int N);
float GPU_big_dot(float *A, float *B, int N);
float *get_random_vector(int N);
long long start_timer();
long long stop_timer(long long start_time, const char *name);
void die(const char *message);

int main(int argc, char **argv) {
	// Seed the random generator (use a constant here for repeatable results)
	srand(10);

	// Determine the vector length
	//int N = 100000;  // default value
	int N = 1 << 24;  // default value
	if (argc > 1) N = atoi(argv[1]); // user-specified value

	// Generate two random vectors
	long long vector_start_time = start_timer();
	float *A = get_random_vector(N);
	float *B = get_random_vector(N);
	stop_timer(vector_start_time, "Vector generation");
	
	// Compute their dot product on the CPU
	long long CPU_start_time = start_timer();
	float C_CPU = CPU_big_dot(A, B, N);
	long long CPU_time = stop_timer(CPU_start_time, "\nCPU");
	
	// Compute their dot product on the GPU
	long long GPU_start_time = start_timer();
	float C_GPU = GPU_big_dot(A, B, N);
	long long GPU_time = stop_timer(GPU_start_time, "\t            Total");
	
	// Compute the speedup or slowdown
	if (GPU_time > CPU_time) printf("\nCPU outperformed GPU by %.2fx\n", (float) GPU_time / (float) CPU_time);
	else                     printf("\nGPU outperformed CPU by %.2fx\n", (float) CPU_time / (float) GPU_time);
	
	// Check the correctness of the GPU results
        if (fabs(C_CPU - C_GPU) > 0.000001) 
	  printf("\nvalues incorrect, CPU dot product = %f, GPU dot product = %f\n", C_CPU, C_GPU);
	else           
	  printf("\nvalues correct, CPU dot product = %f, GPU dot product = %f\n", C_CPU, C_GPU);

}

// A GPU kernel that computes the vector dot product of A and B
// (each thread computes a single value of the result)
__global__ void dot_product_kernel(float *A, float *B, float *C, int N) {
	// Determine which element this thread is computing
	int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	
	// Compute a single element of the result vector (if the element is valid)
	if (thread_id < N) C[thread_id] = A[thread_id] * B[thread_id];
}

// Returns the vector dot product of A and B (computed on the GPU)
float GPU_big_dot(float *A_CPU, float *B_CPU, int N) {
	
	long long memory_start_time = start_timer();

	// Allocate GPU memory for the inputs and the result
	int vector_size = N * sizeof(float);
	float *A_GPU, *B_GPU, *C_GPU;
	if (hipMalloc((void **) &A_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	if (hipMalloc((void **) &B_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	if (hipMalloc((void **) &C_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	
	// Transfer the input vectors to GPU memory
	hipMemcpy(A_GPU, A_CPU, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(B_GPU, B_CPU, vector_size, hipMemcpyHostToDevice);
	
	stop_timer(memory_start_time, "\nGPU:\t  Transfer to GPU");
	
	// Determine the number of thread blocks in the grid 
	int blocks_per_grid = (int) ((float) (N + threads_per_block - 1) / (float) threads_per_block);
	
	// Execute the kernel to compute the vector dot product on the GPU
	long long kernel_start_time = start_timer();
	dot_product_kernel<<< blocks_per_grid , threads_per_block >>> (A_GPU, B_GPU, C_GPU, N);
	hipDeviceSynchronize();  // this is only needed for timing purposes
	stop_timer(kernel_start_time, "\t Kernel execution");
	
	// Check for kernel errors
	hipError_t error = hipGetLastError();
	if (error) {
	  char message[256];
	  sprintf(message, "CUDA error: %s", hipGetErrorString(error));
	  die(message);
	}
	
	// Allocate CPU memory for the result
	float *C_CPU = (float *) malloc(vector_size);
	if (C_CPU == NULL) die("Error allocating CPU memory");
	
	// Transfer the result from the GPU to the CPU
	memory_start_time = start_timer();
	hipMemcpy(C_CPU, C_GPU, vector_size, hipMemcpyDeviceToHost);
	stop_timer(memory_start_time, "\tTransfer from GPU");
	
	// Free the GPU memory
	hipFree(A_GPU);
	hipFree(B_GPU);
	hipFree(C_GPU);

        // Do the summation of multiplication in CPU
        float sum = 0;
	for (int i = 0; i < N; i++) sum += C_CPU[i]; 
	
	return sum;
}

// Returns the vector dot product of A and B
float CPU_big_dot(float *A, float *B, int N) {	
	// Compute the dot product
        float sum = 0;
	for (int i = 0; i < N; i++) sum += A[i] * B[i];
	
	// Return the result
	return sum;
}

// Returns a randomized vector containing N elements
float *get_random_vector(int N) {
	if (N < 1) die("Number of elements must be greater than zero");
	
	// Allocate memory for the vector
	float *V = (float *) malloc(N * sizeof(float));
	if (V == NULL) die("Error allocating CPU memory");
	
	// Populate the vector with random numbers
	for (int i = 0; i < N; i++) V[i] = (float) rand() / (float) rand();
	
	// Return the randomized vector
	return V;
}

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}

// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, const char *name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
	printf("%s: %.5f sec\n", name, ((float) (end_time - start_time)) / (1000 * 1000));
	return end_time - start_time;
}

// Prints the specified message and quits
void die(const char *message) {
	printf("%s\n", message);
	exit(1);
}

